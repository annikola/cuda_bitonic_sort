#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel function to add two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int size = 256;
    int a[size], b[size], c[size];
    int *d_a, *d_b, *d_c;
    hipError_t err, kernelErr;

    // Initialize host arrays
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // for (int i = 0; i < size; i++) {
    //     printf("a[%d] = %d\n", i, a[i]);
    //     printf("b[%d] = %d\n", i, b[i]);
    // }

    // Allocate memory on the GPU
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy data from host to GPU
    err = hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA error during hipMemcpy (d_a): %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA error during hipMemcpy (d_b): %s\n", hipGetErrorString(err));
    }

    // Launch kernel with 256 threads in a single block
    add<<<1, 256>>>(d_a, d_b, d_c, size);

    // Check for kernel launch errors
    kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(kernelErr));
    }

    // Synchronize to ensure kernel execution is complete
    hipDeviceSynchronize();


    // Copy result back to the host
    err = hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA error during hipMemcpy (c_a): %s\n", hipGetErrorString(err));
    }

    // Print the results
    for (int i = 0; i < size; i++) {
        printf("c[%d] = %d\n", i, c[i]);
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
