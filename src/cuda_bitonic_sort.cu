#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel function to add two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int size = 256;
    int a[size], b[size], c[size];
    int *d_a, *d_b, *d_c;

    // Initialize host arrays
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate memory on the GPU
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy data from host to GPU
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 256 threads in a single block
    add<<<1, 256>>>(d_a, d_b, d_c, size);

    // Copy result back to the host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < size; i++) {
        printf("c[%d] = %d\n", i, c[i]);
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
