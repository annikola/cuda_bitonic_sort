#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../include/sorting_helpers.h"

#define MAX_THREADS 1024
#define MAX_LOCAL_ELEMENTS 2048
#define MIN_ARGS 1
#define MIN_Q 11
#define MAX_Q 27

int isAscending(int *A, int n);

__global__ void external_exchanges(int *a, int j, int k) {

    int i, jj, jjj, minmax, tid, dummy, total_threads, total_blocks;

    total_threads = blockDim.x * blockDim.y * blockDim.z;
    total_blocks = gridDim.x * gridDim.y * gridDim.z;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    jj = 1 << j;
    jjj = 2 << k;
    if ((tid & jj) != 0) {
        i = tid + total_threads * total_blocks - jj;
    } else {
        i = tid;
    }
    minmax = i & jjj;
    if (minmax == 0 && a[i] > a[i + jj]) {
        dummy = a[i];
        a[i] = a[i + jj];
        a[i + jj] = dummy;
    }
    if (minmax != 0 && a[i] < a[i + jj]) {
        dummy = a[i];
        a[i] = a[i + jj];
        a[i + jj] = dummy;
    }
}

__global__ void internal_exchanges(int *a, int k, int flow) {

    __shared__ int local_elements[MAX_LOCAL_ELEMENTS];
    int i, i_mod, j, jj, jjj, minmax, ltid, tid, dummy;

    tid = blockIdx.x * blockDim.x + threadIdx.x;
    ltid = threadIdx.x;

    // Initialize the shared memory inside the block (each thread reads two elements...)
    local_elements[ltid] = a[tid + MAX_THREADS * blockIdx.x]; // tid & ((1 << k) − 1)
    local_elements[ltid + MAX_THREADS] = a[tid + MAX_THREADS * (blockIdx.x + 1)];
    __syncthreads();

    for (j = k - 1; j >= 0; j--) {
        jj = 1 << j;
        jjj = 2 << flow;
        if ((tid & jj) != 0) {
            i = tid + MAX_THREADS * (blockIdx.x + 1) - jj;
        } else {
            i = tid + MAX_THREADS * blockIdx.x;
        }
        minmax = i & jjj;
        i_mod = i & ((1 << 11) - 1); // i_mod = i % MAX_LOCAL_ELEMENTS;
        if (minmax == 0 && local_elements[i_mod] > local_elements[i_mod + jj]) {
            dummy = local_elements[i_mod];
            local_elements[i_mod] = local_elements[i_mod + jj];
            local_elements[i_mod + jj] = dummy;
        }
        if (minmax != 0 && local_elements[i_mod] < local_elements[i_mod + jj]) {
            dummy = local_elements[i_mod];
            local_elements[i_mod] = local_elements[i_mod + jj];
            local_elements[i_mod + jj] = dummy;
        }
        __syncthreads();
    }

    a[tid + MAX_THREADS * blockIdx.x] = local_elements[ltid];
    a[tid + MAX_THREADS * (blockIdx.x + 1)] = local_elements[ltid + MAX_THREADS];
}

__global__ void prephase_exchanges(int *a) {

    __shared__ int local_elements[MAX_LOCAL_ELEMENTS];
    int i, i_mod, k, j, jj, jjj, minmax, ltid, tid, dummy;

    tid = blockIdx.x * blockDim.x + threadIdx.x;
    ltid = threadIdx.x;

    // Initialize the shared memory inside the block (each thread reads two elements...)
    local_elements[ltid] = a[tid + MAX_THREADS * blockIdx.x]; // tid & ((1 << k) − 1)
    local_elements[ltid + MAX_THREADS] = a[tid + MAX_THREADS * (blockIdx.x + 1)];
    __syncthreads();

    for (k = 0; k < 11; k++) {
        for (j = k; j >= 0; j--) {
            jj = 1 << j;
            jjj = 2 << k;
            if ((tid & jj) != 0) {
                i = tid + MAX_THREADS * (blockIdx.x + 1) - jj;
            } else {
                i = tid + MAX_THREADS * blockIdx.x;
            }
            minmax = i & jjj;
            i_mod = i & ((1 << 11) - 1); // i_mod = i % MAX_LOCAL_ELEMENTS;
            if (minmax == 0 && local_elements[i_mod] > local_elements[i_mod + jj]) {
                dummy = local_elements[i_mod];
                local_elements[i_mod] = local_elements[i_mod + jj];
                local_elements[i_mod + jj] = dummy;
            }
            if (minmax != 0 && local_elements[i_mod] < local_elements[i_mod + jj]) {
                dummy = local_elements[i_mod];
                local_elements[i_mod] = local_elements[i_mod + jj];
                local_elements[i_mod + jj] = dummy;
            }
            __syncthreads();
        }
    }

    a[tid + MAX_THREADS * blockIdx.x] = local_elements[ltid];
    a[tid + MAX_THREADS * (blockIdx.x + 1)] = local_elements[ltid + MAX_THREADS];
}

int main(int argc, char *argv[]) {

    int i, j, k, Q, A_size, blocks, threads;
    int *A, *d_a;
    float elapsed_time;
    hipEvent_t start, stop;
    hipError_t err;

    if (argc < MIN_ARGS + 1) {
        printf("Missing %d argument(s)\n", MIN_ARGS + 1 - argc);
        return 1;
    }

    Q = atoi(argv[1]);
    if (Q < MIN_Q || Q > MAX_Q) {
        printf("Please insert a value for Q between %d and %d\n", MIN_Q, MAX_Q);
        return 1;
    }

    A_size = 1 << Q;
    A = (int *)malloc(A_size * sizeof(int));
    for (i = 0; i < A_size; i++) {
        A[i] = rand();
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); // Start the timing...

    hipMalloc((void **)&d_a, A_size * sizeof(int));

    err = hipMemcpy(d_a, A, A_size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA error during hipMemcpy (d_a): %s\n", hipGetErrorString(err));
    }

    if (A_size < MAX_THREADS) {
        blocks = 1;
        threads = A_size;
    } else {
        blocks = (A_size / MAX_THREADS) / 2;
        threads = MAX_THREADS;
    }

    prephase_exchanges<<<blocks, threads>>>(d_a);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error during prephase_exchanges: %s\n", hipGetErrorString(err));
    }

    for (k = 11; k < Q; k++) {
        for (j = k; j > 10; j--) {
            external_exchanges<<<blocks, threads>>>(d_a, j, k);
            err = hipGetLastError();
            if (err != hipSuccess) {
                printf("CUDA error during external_exchanges: %s\n", hipGetErrorString(err));
            }
        }
        internal_exchanges<<<blocks, threads>>>(d_a, 11, k);
        if (err != hipSuccess) {
            printf("CUDA error during internal_exchanges: %s\n", hipGetErrorString(err));
        }
    }

    err = hipMemcpy(A, d_a, A_size * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA error during hipMemcpy (B_a): %s\n", hipGetErrorString(err));
    }

    hipEventRecord(stop, 0); // Stop the timing...
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Total execution time: %f ms\n", elapsed_time);

    // Check the validity of the results
    if (isAscending(A, A_size)) {
        printf("Correctly sorted!\n");
    } else {
        printf("Falsely sorted!\n");
    }

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);

    return 0;
}

int isAscending(int *A, int n) {

    int i;

    for (i = 0; i < n - 1; i++) {
        if (A[i] > A[i + 1]) {
            return 0;
        }
    }

    return 1;
}
